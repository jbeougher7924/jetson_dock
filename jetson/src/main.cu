#include <iostream>
#include <hip/hip_runtime.h>

// CUDA Kernel function to print "Hello, CUDA!" from each thread
__global__ void helloCUDA()
{
    printf("Hello, CUDA! from thread %d\n", threadIdx.x);
}

int main()
{
    // Launch the helloCUDA kernel with 1 block and 10 threads per block
    helloCUDA<<<1, 10>>>();

    // Wait for the GPU to finish
    hipDeviceSynchronize();
	
	hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
        return 1;
    }

    std::cout << "CUDA kernel executed successfully!" << std::endl;

    // Print a message from the CPU
    std::cout << "Hello from CPU!" << std::endl;

    return 0;
}
