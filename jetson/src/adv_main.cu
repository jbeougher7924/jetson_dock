#include <iostream>
#include <hip/hip_runtime.h>

// CUDA Kernel function to add two arrays element-wise
__global__ void addArrays(float *a, float *b, float *c, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    const int N = 1000; // Array size
    const int blockSize = 256;
    const int numBlocks = (N + blockSize - 1) / blockSize;

    // Allocate memory on the host (CPU)
    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_c = new float[N];

    // Initialize input arrays on the host
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Allocate memory on the device (GPU)
    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N * sizeof(float));
    hipMalloc((void **)&d_b, N * sizeof(float));
    hipMalloc((void **)&d_c, N * sizeof(float));

    // Copy input arrays from host to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernel to add arrays
    addArrays<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Copy result array from device to host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result array
    std::cout << "Resultant array:" << std::endl;
    for (int i = 0; i < N; i++)
    {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free allocated memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free allocated memory on the host
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
